#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define TILE_WIDTH 16

bool InitCUDA(){
	int count;
	hipDeviceProp_t prop;

	hipGetDeviceCount(&count);
	if(count == 0){
		fprintf(stderr, "There is no device.\n");
		return false;
	}

	int i;
	for(i = 0; i < count; i++){
		if(hipGetDeviceProperties(&prop, i) == hipSuccess){
			if(prop.major >= 1){
				break;
			}
		}
	} 

	if(i == count){
		fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
		return false;
	}

	hipSetDevice(i);
	return true;
}

void matgen(float* a, int n){
	int i, j;
	for(i = 0; i < n; i++){
		for(j = 0; j < n; j++){
			a[i * n + j] = (float)rand();
		}
	}
}	
__global__ void MatrixMulKernel(float* Md, float* Nd, float* Pd, int Width){
	
	int tx = blockIdx.x * TILE_WIDTH + threadIdx.x;
	int ty = blockIdx.y * TILE_WIDTH + threadIdx.y;
	float Pvalue = 0;

	for(int k = 0; k < Width; k++){
		float Mdelement = Md[ty * Width + k];
		float Ndelement = Nd[k * Width + tx];
		Pvalue += Mdelement * Ndelement;
	}
	Pd[ty * Width+ tx] = Pvalue;
}


int main(){
    FILE *f = fopen("result.txt", "w");
        if (f == NULL)
        {
          printf("Error opening file!\n");
          exit(1);
         }

	if(!InitCUDA())
		return 0;
	printf("CUDA initialized.\n");
    fprintf(f, "CUDA initialized.\n", end);


	int i;
    for(i = 8; i <= 1024; i *= 2){
    clock_t start = clock();
	float* M, *N, *P;
    M = (float*) malloc(sizeof(float) * i * i);
    N = (float*) malloc(sizeof(float) * i * i);
    P = (float*) malloc(sizeof(float) * i * i);
	
	srand(0);
	matgen(M, i);
	matgen(N, i);

	int size = i * i * sizeof(float);
	float* Md, *Nd, *Pd;

	hipMalloc((void**) &Md, size);
	hipMemcpy(Md, M, size, hipMemcpyHostToDevice);
	hipMalloc((void**) &Nd, size);
	hipMemcpy(Nd, N, size, hipMemcpyHostToDevice);
	hipMalloc((void**) &Pd, size);

	dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
	dim3 dimGrid(ceil(float(i)/TILE_WIDTH), ceil(float(i)/TILE_WIDTH));
    MatrixMulKernel<<<dimGrid, dimBlock>>>(Md, Nd, Pd, i);

	hipMemcpy(P, Pd, size, hipMemcpyDeviceToHost);
	hipFree(Md);
	hipFree(Nd);
	hipFree(Pd);

	clock_t end = (clock() - start) / 1000;
	printf("%d * %d, uses time: %ldms\n", i, i, end);
	fprintf(f, "%d * %d, uses time: %ldms\n", i, i, end);
    }

    fclose(f);
	return 0;
}
